#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/driver_types.h>
#include <hiprand.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "support.h"
#include "hash_kernel.cu"
#include "nonce_kernel.cu"
#include "reduction_kernel.cu"

// to activate debug statements
#define DEBUG 1

// program constants
#define BLOCK_SIZE 1024
#define SEED       123

// solution constants
#define MAX     123123123
#define TARGET  20

// functions used
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions);
void err_check(hipError_t ret, char* msg, int exit_code);


/* Main ------------------ //
*   This is the main program.
*/
int main(int argc, char* argv[]) {

    // Catch console errors
    if (argc != 6) {
        printf("USE LIKE THIS: gpu_mining transactions.csv n_transactions trials out.csv time.csv\n");
        return EXIT_FAILURE;
    }


    // Output files
    FILE* output_file = fopen(argv[4], "w");
    FILE* time_file   = fopen(argv[5], "w");

    // Read in the transactions
    unsigned int n_transactions = strtoul(argv[2], NULL, 10);
    unsigned int* transactions = (unsigned int*)calloc(n_transactions, sizeof(unsigned int));
    read_file(argv[1], transactions, n_transactions);

    // get the number of trials
    unsigned int trials = strtoul(argv[3], NULL, 10);


    // -------- Start Mining ------------------------------------------------------- //
    // ----------------------------------------------------------------------------- //
    
    // Set timer and cuda error return
    Timer timer;
    startTime(&timer);
    hipError_t cuda_ret;

    // To use with kernels
    int num_blocks = ceil((float)trials / (float)BLOCK_SIZE);
    dim3 dimGrid(num_blocks, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);


    // ------ Step 1: generate the nonce values ------ //

    // Allocate the nonce device memory
    unsigned int* device_nonce_array;
    cuda_ret = hipMalloc((void**)&device_nonce_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate nonces to device memory!", 1);

    // Launch the nonce kernel
    nonce_kernel <<< dimGrid, dimBlock >>> (
        device_nonce_array, // put nonces into here
        trials,             // size of array
        MAX,                // to mod with
        SEED                // random seed
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch nonce kernel!", 2);

    // Get nonces from device memory
    unsigned int* nonce_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(nonce_array, device_nonce_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read nonce from device memory!", 3);


    // ------ Step 2: Generate the hash values ------ //

    // Allocate the hash device memory
    unsigned int* device_hash_array;
    cuda_ret = hipMalloc((void**)&device_hash_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate hashes to device memory!", 1);

    // Allocate the transactions device memory
    unsigned int* device_transactions;
    cuda_ret = hipMalloc((void**)&device_transactions, n_transactions * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate transactions to device memory!", 1);
    cuda_ret = hipMemcpy(device_transactions, transactions, n_transactions * sizeof(unsigned int), hipMemcpyHostToDevice);
    err_check(cuda_ret, (char*)"Unable to copy transactions from host memory to device memory!", 3);

    // Launch the hash kernel
    hash_kernel <<< dimGrid, dimBlock >>> (
        device_hash_array,  // put hashes into here
        device_nonce_array, // nonce values
        trials,             // size of array
        device_transactions,// transactions made
        n_transactions,     // number of transactions
        MAX                 // to mod with
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch hash kernel!", 2);

    // Get hashes from device memory
    unsigned int* hash_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(hash_array, device_hash_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read hash from device memory!", 3);

    // Free memory
    free(transactions);
    free(nonce_array);
    free(hash_array);
    hipFree(device_transactions);


    // ------ Step 3: Find the nonce with the minimum hash value ------ //

    // Calculate the minimum array size
    unsigned int min_array_size = ceil(trials/(float)(2.0*BLOCK_SIZE));

    // Allocate the min hash array device memory
    unsigned int* device_min_hash_array;
    cuda_ret = hipMalloc((void**)&device_min_hash_array, min_array_size * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate min hashes to device memory!", 1);

    // Allocate the min nonce array device memory
    unsigned int* device_min_nonce_array;
    cuda_ret = hipMalloc((void**)&device_min_nonce_array, min_array_size * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate min nonces to device memory!", 1);
    
    // Launch the reduction kernel
    dimGrid.x = min_array_size;
    reduction_kernel <<< dimGrid, dimBlock >>> (
        device_hash_array,      // hash values
        device_nonce_array,     // nonce values
        trials,                 // size of arrays
        device_min_hash_array,  // put min hashes into here
        device_min_nonce_array, // put nonces into here
        MAX                     // max value
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch reduction kernel!", 2);

    // Get the min hashes from device memory
    unsigned int* min_hash_array = (unsigned int*)calloc(min_array_size, sizeof(unsigned int));
    cuda_ret = hipMemcpy(min_hash_array, device_min_hash_array, min_array_size * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read min hashes from device memory!", 3);
    
    // Get the nonces from device memory
    unsigned int* min_nonce_array = (unsigned int*)calloc(min_array_size, sizeof(unsigned int));
    cuda_ret = hipMemcpy(min_nonce_array, device_min_nonce_array, min_array_size * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read min nonces from device memory!", 3);

    // Get the nonce with the min hash from device memory
    unsigned int min_hash = MAX;
    unsigned int min_nonce = MAX;
    for (int i = 0; i < min_array_size; i++) {
        if (min_hash_array[i] < min_hash) {
            min_hash  = min_hash_array[i];
            min_nonce = min_nonce_array[i];
        }
    }

    // Free memory
    free(min_nonce_array);
    free(min_hash_array);
    hipFree(device_nonce_array);
    hipFree(device_hash_array);
    hipFree(device_min_nonce_array);
    hipFree(device_min_hash_array);

    stopTime(&timer);
    // ----------------------------------------------------------------------------- //
    // -------- Finish Mining ------------------------------------------------------ //


    // Get if suceeded
    char* res = (char*)malloc(8 * sizeof(char));
    if (min_hash < TARGET)  res = (char*)"Success!";
    else                    res = (char*)"Failure.";

    // Show results in console
    if (DEBUG) 
        printf("%s\n   Min hash:  %u\n   Min nonce: %u\n   %f seconds\n",
            res,
            min_hash,
            min_nonce,
            elapsedTime(timer)
        );

    // Print results
    fprintf(output_file, "%s\n%u\n%u\n", res, min_hash, min_nonce);
    fprintf(time_file, "%f\n", elapsedTime(timer));

    // Cleanup
    fclose(time_file);
    fclose(output_file);

    return 0;
} // End Main -------------------------------------------- //



/* Read File -------------------- //
*   Reads in a file of transactions. 
*/
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions) {

    // open file
    FILE* trans_file = fopen(file, "r");
    if (trans_file == NULL)
        fprintf(stderr, "ERROR: could not read the transaction file.\n"),
        exit(-1);

    // read items
    char line[100] = { 0 };
    for (int i = 0; i < n_transactions && fgets(line, 100, trans_file); ++i) {
        char* p;
        transactions[i] = strtof(line, &p);
    }

    fclose(trans_file);

} // End Read File ------------- //



/* Error Check ----------------- //
*   Exits if there is a CUDA error.
*/
void err_check(hipError_t ret, char* msg, int exit_code) {
    if (ret != hipSuccess)
        fprintf(stderr, "%s \"%s\".\n", msg, hipGetErrorString(ret)),
        exit(exit_code);
} // End Error Check ----------- //
